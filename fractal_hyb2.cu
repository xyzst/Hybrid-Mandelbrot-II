#include "hip/hip_runtime.h"
/*
Fractal code for CS 4380 / CS 5351

Copyright (c) 2016, Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is not permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" 
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE 
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE 
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE 
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR 
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF 
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN 
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) 
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
Co-Author: Darren Rambaud
*/

#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "cs43805351.h"

static const int ThreadsPerBlock = 512;

static const double Delta = 0.005491;
static const double xMid = 0.745796;
static const double yMid = 0.105089;

static __global__
void FractalKernel(const int from_frame, const int to_frame, \
                    const int width, unsigned char pic_d[])
{
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // if idx falls within the range between from_frame && to_frame
    if (idx <= to_frame * width * width) {
        const int col = idx % width;
        const int row = (idx / width) % width;
        const int frame = idx / (width * width) + from_frame;

        const double myDelta = Delta * pow(0.99, frame + 1);
        const double xMin = xMid - myDelta;
        const double yMin = yMid - myDelta;
        const double dw = 2.0 * myDelta / width;
        
        const double cy = -yMin - row * dw;
        const double cx = -xMin - col * dw;

        double x = cx;
        double y = cy;
        int depth = 256;
        double x2,
               y2;
        do {
            x2 = x * x;
            y2 = y * y;
            y = 2 * x * y + cy;
            x = x2 - y2 + cx;
            --depth;
        } while((depth > 0) && ((x2 + y2) < 5.0));
        pic_d[(frame - from_frame) * width * width + row * width + col] \
        = (unsigned char)depth; 
    }
}

unsigned char* GPU_Init(const int size)
{
    unsigned char* legerdemain_pic;

    if (hipSuccess != hipMalloc((void **)&legerdemain_pic, size)) {
        fprintf(stderr, "could not allocate memory on GPU\n");
        exit(-1);
    }

    return legerdemain_pic; 
}

void GPU_Exec(const int from_frame, const int to_frame, const int width, \
                unsigned char pic_d[])
{
    FractalKernel<<<((to_frame - from_frame) * width * width + \
     (ThreadsPerBlock - 1)) / ThreadsPerBlock, \
       ThreadsPerBlock>>>(from_frame, to_frame, width, pic_d);
}

void GPU_Fini(const int size, unsigned char pic[], unsigned char pic_d[])
{
    if (hipSuccess != hipMemcpy(pic, pic_d, size, \
      hipMemcpyDeviceToHost)) {
        fprintf(stderr, "copying from device failed\n");
        exit(-1);
    }

    hipFree(pic_d);
}
